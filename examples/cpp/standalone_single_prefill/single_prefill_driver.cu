#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>

#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/prefill.cuh>
#include <flashinfer/attention/variant_helper.cuh>
#include <flashinfer/attention/variants.cuh>
#include <flashinfer/fastdiv.cuh>
#include <flashinfer/layout.cuh>
#include <flashinfer/math.cuh>
#include <flashinfer/pos_enc.cuh>
#include <flashinfer/utils.cuh>
#include <iomanip>
#include <iostream>
#include <optional>
#include <string>
#include <vector>

namespace flashinfer
{

// Parameter struct for SinglePrefill
template <typename DTypeOs, typename IdTypes> struct SinglePrefillParams
{
    using DTypeQ = half;
    using DTypeKV = half;
    using DTypeO = DTypeOs;
    using IdType = IdTypes;

    half *q;
    half *k;
    half *v;
    DTypeO *o;
    float *lse;
    uint_fastdiv group_size;

    uint8_t *maybe_custom_mask;
    float *maybe_alibi_slopes;
    double logits_soft_cap;
    double sm_scale;
    double rope_rcp_scale;
    double rope_rcp_theta;

    uint32_t qo_len;
    uint32_t kv_len;
    uint32_t num_qo_heads;
    uint32_t num_kv_heads;
    uint32_t q_stride_n;
    uint32_t q_stride_h;
    uint32_t k_stride_n;
    uint32_t k_stride_h;
    uint32_t v_stride_n;
    uint32_t v_stride_h;
    uint32_t head_dim;
    int32_t window_left;

    bool partition_kv;

    __host__ __device__ __forceinline__ uint32_t
    get_qo_len(uint32_t batch_idx) const
    {
        return qo_len;
    }

    __host__ __device__ __forceinline__ uint32_t
    get_kv_len(uint32_t batch_idx) const
    {
        return kv_len;
    }
};

} // namespace flashinfer

// CPU reference implementation for validation
namespace reference
{

template <typename T>
std::vector<T> single_mha(const std::vector<T> &q,
                          const std::vector<T> &k,
                          const std::vector<T> &v,
                          size_t qo_len,
                          size_t kv_len,
                          size_t num_qo_heads,
                          size_t num_kv_heads,
                          size_t head_dim,
                          bool causal,
                          flashinfer::QKVLayout kv_layout,
                          flashinfer::PosEncodingMode pos_encoding_mode,
                          float rope_scale = 1.0f,
                          float rope_theta = 10000.0f)
{
    float sm_scale = 1.0f / std::sqrt(static_cast<float>(head_dim));
    std::vector<T> o(qo_len * num_qo_heads * head_dim, static_cast<T>(0.0f));
    std::vector<float> att(kv_len);
    size_t group_size = num_qo_heads / num_kv_heads;

    for (size_t qo_head_idx = 0; qo_head_idx < num_qo_heads; ++qo_head_idx) {
        size_t kv_head_idx = qo_head_idx / group_size;

        for (size_t q_idx = 0; q_idx < qo_len; ++q_idx) {
            // 1. Compute attention scores
            float max_val = -5e4f;

            for (size_t kv_idx = 0; kv_idx < kv_len; ++kv_idx) {
                if (causal && kv_idx > kv_len + q_idx - qo_len) {
                    att[kv_idx] = -5e4f;
                    continue;
                }

                // Compute dot product between Q and K
                float score = 0.0f;
                for (size_t d = 0; d < head_dim; ++d) {
                    float q_val = 0.0f;
                    float k_val = 0.0f;

                    // Get Q value - always NHD layout
                    size_t q_offset = q_idx * num_qo_heads * head_dim +
                                      qo_head_idx * head_dim + d;
                    q_val = static_cast<float>(q[q_offset]);

                    // Get K value - depends on layout
                    if (kv_layout == flashinfer::QKVLayout::kNHD) {
                        size_t k_offset = kv_idx * num_kv_heads * head_dim +
                                          kv_head_idx * head_dim + d;
                        k_val = static_cast<float>(k[k_offset]);
                    }
                    else {
                        size_t k_offset = kv_head_idx * kv_len * head_dim +
                                          kv_idx * head_dim + d;
                        k_val = static_cast<float>(k[k_offset]);
                    }

                    score += q_val * k_val;
                }
                score *= sm_scale;

                att[kv_idx] = score;
                max_val = std::max(max_val, score);
            }

            // 2. Apply softmax
            float sum_exp = 0.0f;
            for (size_t kv_idx = 0; kv_idx < kv_len; ++kv_idx) {
                if (causal && kv_idx > kv_len + q_idx - qo_len) {
                    att[kv_idx] = 0.0f;
                }
                else {
                    att[kv_idx] = std::exp(att[kv_idx] - max_val);
                    sum_exp += att[kv_idx];
                }
            }

            // Normalize
            for (size_t kv_idx = 0; kv_idx < kv_len; ++kv_idx) {
                if (sum_exp > 0.0f) {
                    att[kv_idx] /= sum_exp;
                }
            }

            // 3. Compute weighted sum of values
            for (size_t d = 0; d < head_dim; ++d) {
                float weighted_sum = 0.0f;

                for (size_t kv_idx = 0; kv_idx < kv_len; ++kv_idx) {
                    float v_val = 0.0f;

                    // Get V value - depends on layout
                    if (kv_layout == flashinfer::QKVLayout::kNHD) {
                        size_t v_offset = kv_idx * num_kv_heads * head_dim +
                                          kv_head_idx * head_dim + d;
                        v_val = static_cast<float>(v[v_offset]);
                    }
                    else {
                        size_t v_offset = kv_head_idx * kv_len * head_dim +
                                          kv_idx * head_dim + d;
                        v_val = static_cast<float>(v[v_offset]);
                    }

                    weighted_sum += att[kv_idx] * v_val;
                }

                // Store result in output
                size_t o_offset = q_idx * num_qo_heads * head_dim +
                                  qo_head_idx * head_dim + d;
                o[o_offset] = static_cast<T>(weighted_sum);
            }
        }
    }

    return o;
}

} // namespace reference

// Function to validate GPU results against CPU reference
bool validate_results(const thrust::host_vector<half> &gpu_output,
                      const std::vector<half> &cpu_output,
                      float rtol = 1e-3f,
                      float atol = 1e-3f)
{
    if (gpu_output.size() != cpu_output.size()) {
        std::cerr << "Size mismatch: GPU=" << gpu_output.size()
                  << " vs CPU=" << cpu_output.size() << std::endl;
        return false;
    }

    int errors = 0;
    float max_diff = 0.0f;
    float max_rel_diff = 0.0f;

    for (size_t i = 0; i < gpu_output.size(); ++i) {
        float gpu_val = static_cast<float>(gpu_output[i]);
        float cpu_val = static_cast<float>(cpu_output[i]);
        float abs_diff = std::abs(gpu_val - cpu_val);
        float rel_diff =
            (cpu_val != 0.0f) ? abs_diff / std::abs(cpu_val) : abs_diff;

        max_diff = std::max(max_diff, abs_diff);
        max_rel_diff = std::max(max_rel_diff, rel_diff);

        bool close = (abs_diff <= atol + rtol * std::abs(cpu_val));
        if (!close) {
            errors++;
            if (errors <= 10) { // Print just a few examples
                std::cerr << "Mismatch at " << i << ": GPU=" << gpu_val
                          << " CPU=" << cpu_val << " (diff=" << abs_diff << ")"
                          << std::endl;
            }
        }
    }

    float error_rate = static_cast<float>(errors) / gpu_output.size();
    std::cout << "\nValidation Results:" << std::endl;
    std::cout << "  Max absolute difference: " << max_diff << std::endl;
    std::cout << "  Max relative difference: " << max_rel_diff << std::endl;
    std::cout << "  Error rate: " << (error_rate * 100) << "% (" << errors
              << " / " << gpu_output.size() << " elements)" << std::endl;
    std::cout << "  Status: " << (error_rate < 0.05 ? "PASSED" : "FAILED")
              << std::endl;

    // Allow up to 5% error rate (similar to the threshold used in the unit
    // tests)
    return error_rate < 0.05;
}

using namespace flashinfer;

// Helper class to convert strings to parameters
class ArgParser
{
public:
    static bool get_bool(const char *arg, bool default_val)
    {
        return arg == nullptr
                   ? default_val
                   : (std::string(arg) == "1" || std::string(arg) == "true");
    }

    static int get_int(const char *arg, int default_val)
    {
        return arg == nullptr ? default_val : std::atoi(arg);
    }

    static float get_float(const char *arg, float default_val)
    {
        return arg == nullptr ? default_val : std::atof(arg);
    }

    static PosEncodingMode get_pos_encoding_mode(const char *arg)
    {
        if (arg == nullptr)
            return PosEncodingMode::kNone;
        std::string str_val = arg;
        if (str_val == "none")
            return PosEncodingMode::kNone;
        if (str_val == "rope")
            return PosEncodingMode::kRoPELlama;
        if (str_val == "alibi")
            return PosEncodingMode::kALiBi;
        return PosEncodingMode::kNone;
    }

    static QKVLayout get_layout(const char *arg)
    {
        if (arg == nullptr)
            return QKVLayout::kNHD;
        std::string str_val = arg;
        if (str_val == "nhd")
            return QKVLayout::kNHD;
        if (str_val == "hnd")
            return QKVLayout::kHND;
        return QKVLayout::kNHD;
    }
};

// Helper function to generate random data on device
void generate_random_data(thrust::device_vector<half> &data,
                          float min_val = -1.0f,
                          float max_val = 1.0f)
{
    thrust::host_vector<half> host_data(data.size());

    thrust::default_random_engine rng(42); // Fixed seed for reproducibility
    thrust::uniform_real_distribution<float> dist(min_val, max_val);

    for (size_t i = 0; i < host_data.size(); ++i) {
        host_data[i] = static_cast<half>(dist(rng));
    }

    data = host_data;
}

// Dispatch function for half precision
hipError_t dispatch_single_prefill(half *q_ptr,
                                    half *k_ptr,
                                    half *v_ptr,
                                    half *o_ptr,
                                    half *tmp_ptr,
                                    float *lse_ptr,
                                    uint32_t num_qo_heads,
                                    uint32_t num_kv_heads,
                                    uint32_t qo_len,
                                    uint32_t kv_len,
                                    uint32_t head_dim,
                                    QKVLayout kv_layout,
                                    PosEncodingMode pos_encoding_mode,
                                    bool causal,
                                    bool use_fp16_qk_reduction,
                                    double sm_scale,
                                    int32_t window_left,
                                    double rope_scale,
                                    double rope_theta,
                                    hipStream_t stream)
{
    // Compute strides based on layout
    uint32_t q_stride_n = num_qo_heads * head_dim;
    uint32_t q_stride_h = head_dim;
    uint32_t k_stride_n, k_stride_h, v_stride_n, v_stride_h;

    if (kv_layout == QKVLayout::kNHD) {
        k_stride_n = num_kv_heads * head_dim;
        k_stride_h = head_dim;
        v_stride_n = num_kv_heads * head_dim;
        v_stride_h = head_dim;
    }
    else {
        k_stride_h = kv_len * head_dim;
        k_stride_n = head_dim;
        v_stride_h = kv_len * head_dim;
        v_stride_n = head_dim;
    }

    // Configure mask mode
    const MaskMode mask_mode = causal ? MaskMode::kCausal : MaskMode::kNone;

    // Constants for prefill kernel
    constexpr uint32_t HEAD_DIM_QK = 128;
    constexpr uint32_t HEAD_DIM_VO = 128;
    constexpr PosEncodingMode POS_ENCODING_MODE = PosEncodingMode::kRoPELlama;
    constexpr bool USE_FP16_QK_REDUCTION = false;

    hipError_t status = hipSuccess;

    if (causal) {
        // Causal attention
        using AttentionVariantType =
            DefaultAttention<false, false, false, false>;
        using Params = SinglePrefillParams<half, int32_t>;

        Params params;
        params.q = q_ptr;
        params.k = k_ptr;
        params.v = v_ptr;
        params.o = o_ptr;
        params.lse = lse_ptr;
        params.num_qo_heads = num_qo_heads;
        params.num_kv_heads = num_kv_heads;
        params.group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
        params.qo_len = qo_len;
        params.kv_len = kv_len;
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.k_stride_n = k_stride_n;
        params.k_stride_h = k_stride_h;
        params.v_stride_n = v_stride_n;
        params.v_stride_h = v_stride_h;
        params.head_dim = head_dim;
        params.window_left = window_left;
        params.partition_kv = false;
        params.maybe_custom_mask = nullptr;
        params.maybe_alibi_slopes = nullptr;
        params.logits_soft_cap = 0.0;
        params.sm_scale = sm_scale;
        params.rope_rcp_scale = 1.0 / rope_scale;
        params.rope_rcp_theta = 1.0 / rope_theta;

        status = SinglePrefillWithKVCacheDispatched<
            HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE, USE_FP16_QK_REDUCTION,
            MaskMode::kCausal, AttentionVariantType>(params, tmp_ptr, stream);
    }
    else {
        // Non-causal attention
        using AttentionVariantType =
            DefaultAttention<false, false, false, false>;
        using Params = SinglePrefillParams<half, int32_t>;

        Params params;
        params.q = q_ptr;
        params.k = k_ptr;
        params.v = v_ptr;
        params.o = o_ptr;
        params.lse = lse_ptr;
        params.num_qo_heads = num_qo_heads;
        params.num_kv_heads = num_kv_heads;
        params.group_size = uint_fastdiv(num_qo_heads / num_kv_heads);
        params.qo_len = qo_len;
        params.kv_len = kv_len;
        params.q_stride_n = q_stride_n;
        params.q_stride_h = q_stride_h;
        params.k_stride_n = k_stride_n;
        params.k_stride_h = k_stride_h;
        params.v_stride_n = v_stride_n;
        params.v_stride_h = v_stride_h;
        params.head_dim = head_dim;
        params.window_left = window_left;
        params.partition_kv = false;
        params.maybe_custom_mask = nullptr;
        params.maybe_alibi_slopes = nullptr;
        params.logits_soft_cap = 0.0;
        params.sm_scale = sm_scale;
        params.rope_rcp_scale = 1.0 / rope_scale;
        params.rope_rcp_theta = 1.0 / rope_theta;

        status = SinglePrefillWithKVCacheDispatched<
            HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE, USE_FP16_QK_REDUCTION,
            MaskMode::kNone, AttentionVariantType>(params, tmp_ptr, stream);
    }

    return status;
}

// Function to calculate FLOPs for single_prefill
double calculate_flops(uint32_t qo_len,
                       uint32_t kv_len,
                       uint32_t num_qo_heads,
                       uint32_t head_dim,
                       bool causal)
{
    double flops;
    if (causal) {
        // For causal attention: qo_len * (2 * kv_len - qo_len) * 2 *
        // num_qo_heads * head_dim
        flops = static_cast<double>(qo_len) * (2.0 * kv_len - qo_len) * 2.0 *
                num_qo_heads * head_dim;
    }
    else {
        // For non-causal attention: qo_len * kv_len * 4 * num_qo_heads *
        // head_dim
        flops = static_cast<double>(qo_len) * kv_len * 4.0 * num_qo_heads *
                head_dim;
    }
    return flops;
}

void print_usage(const char *program_name)
{
    std::cerr
        << "Usage: " << program_name << " [options]\n"
        << "Options:\n"
        << "  --qo_len <int>            : Query sequence length (default: "
           "512)\n"
        << "  --kv_len <int>            : Key/value sequence length (default: "
           "512)\n"
        << "  --num_qo_heads <int>      : Number of query heads (default: 32)\n"
        << "  --num_kv_heads <int>      : Number of key/value heads (default: "
           "32)\n"
        << "  --head_dim <int>          : Head dimension (default: 128)\n"
        << "  --layout <nhd|hnd>        : KV tensor layout (default: nhd)\n"
        << "  --pos_encoding <none|rope|alibi> : Position encoding mode "
           "(default: none)\n"
        << "  --causal <0|1>            : Use causal mask (default: 1)\n"
        << "  --use_fp16_qk <0|1>       : Use FP16 for QK reduction (default: "
           "0)\n"
        << "  --window_left <int>       : Window left size (default: -1)\n"
        << "  --rope_scale <float>      : RoPE scale factor (default: 1.0)\n"
        << "  --rope_theta <float>      : RoPE theta (default: 10000.0)\n"
        << "  --iterations <int>        : Number of iterations for timing "
           "(default: 10)\n"
        << "  --warmup <int>            : Number of warmup iterations "
           "(default: 5)\n"
        << "  --validate <0|1>          : Validate against CPU reference "
           "(default: 0)\n";
}

int main(int argc, char *argv[])
{
    // Default parameter values
    uint32_t qo_len = 512;
    uint32_t kv_len = 512;
    uint32_t num_qo_heads = 32;
    uint32_t num_kv_heads = 32;
    uint32_t head_dim = 128;
    bool causal = true;
    bool use_fp16_qk_reduction = false;
    QKVLayout kv_layout = QKVLayout::kNHD;
    PosEncodingMode pos_encoding_mode = PosEncodingMode::kNone;
    int32_t window_left = -1;
    float rope_scale = 1.0f;
    float rope_theta = 10000.0f;
    int iterations = 10;
    int warmup = 5;
    bool validate = false;

    // Parse command-line arguments
    for (int i = 1; i < argc; i++) {
        std::string arg = argv[i];
        if (arg == "--qo_len" && i + 1 < argc)
            qo_len = std::atoi(argv[++i]);
        else if (arg == "--kv_len" && i + 1 < argc)
            kv_len = std::atoi(argv[++i]);
        else if (arg == "--num_qo_heads" && i + 1 < argc)
            num_qo_heads = std::atoi(argv[++i]);
        else if (arg == "--num_kv_heads" && i + 1 < argc)
            num_kv_heads = std::atoi(argv[++i]);
        else if (arg == "--head_dim" && i + 1 < argc)
            head_dim = std::atoi(argv[++i]);
        else if (arg == "--causal" && i + 1 < argc)
            causal = ArgParser::get_bool(argv[++i], true);
        else if (arg == "--use_fp16_qk" && i + 1 < argc)
            use_fp16_qk_reduction = ArgParser::get_bool(argv[++i], false);
        else if (arg == "--layout" && i + 1 < argc)
            kv_layout = ArgParser::get_layout(argv[++i]);
        else if (arg == "--pos_encoding" && i + 1 < argc)
            pos_encoding_mode = ArgParser::get_pos_encoding_mode(argv[++i]);
        else if (arg == "--window_left" && i + 1 < argc)
            window_left = std::atoi(argv[++i]);
        else if (arg == "--rope_scale" && i + 1 < argc)
            rope_scale = std::atof(argv[++i]);
        else if (arg == "--rope_theta" && i + 1 < argc)
            rope_theta = std::atof(argv[++i]);
        else if (arg == "--iterations" && i + 1 < argc)
            iterations = std::atoi(argv[++i]);
        else if (arg == "--warmup" && i + 1 < argc)
            warmup = std::atoi(argv[++i]);
        else if (arg == "--validate" && i + 1 < argc)
            validate = ArgParser::get_bool(argv[++i], false);
        else if (arg == "--help") {
            print_usage(argv[0]);
            return 0;
        }
    }

    // Verify that num_qo_heads is divisible by num_kv_heads
    if (num_qo_heads % num_kv_heads != 0) {
        std::cerr << "Error: num_qo_heads must be divisible by num_kv_heads"
                  << std::endl;
        return 1;
    }

    // Display configuration
    std::cout << "Configuration:" << std::endl;
    std::cout << "  qo_len = " << qo_len << std::endl;
    std::cout << "  kv_len = " << kv_len << std::endl;
    std::cout << "  num_qo_heads = " << num_qo_heads << std::endl;
    std::cout << "  num_kv_heads = " << num_kv_heads << std::endl;
    std::cout << "  head_dim = " << head_dim << std::endl;
    std::cout << "  kv_layout = "
              << (kv_layout == QKVLayout::kNHD ? "NHD" : "HND") << std::endl;
    std::cout << "  causal = " << (causal ? "true" : "false") << std::endl;
    std::cout << "  data_type = half" << std::endl;
    std::cout << "  use_fp16_qk_reduction = "
              << (use_fp16_qk_reduction ? "true" : "false") << std::endl;
    std::cout << "  validate = " << (validate ? "true" : "false") << std::endl;

    // Initialize CUDA and create stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocate device memory using Thrust - only for half precision
    thrust::device_vector<half> q(qo_len * num_qo_heads * head_dim);
    thrust::device_vector<half> k(kv_len * num_kv_heads * head_dim);
    thrust::device_vector<half> v(kv_len * num_kv_heads * head_dim);
    thrust::device_vector<half> o(qo_len * num_qo_heads * head_dim);
    thrust::device_vector<half> tmp(qo_len * num_qo_heads * head_dim);
    thrust::device_vector<float> lse(qo_len * num_qo_heads);

    // Generate random data
    generate_random_data(q);
    generate_random_data(k);
    generate_random_data(v);
    thrust::fill(o.begin(), o.end(), half(0.0f));
    thrust::fill(tmp.begin(), tmp.end(), half(0.0f));
    thrust::fill(lse.begin(), lse.end(), 0.0f);

    // Calculate SM scale if not provided
    float sm_scale = 1.0f / std::sqrt(static_cast<float>(head_dim));

    // Warm-up runs
    for (int i = 0; i < warmup; ++i) {
        hipError_t status = dispatch_single_prefill(
            thrust::raw_pointer_cast(q.data()),
            thrust::raw_pointer_cast(k.data()),
            thrust::raw_pointer_cast(v.data()),
            thrust::raw_pointer_cast(o.data()),
            thrust::raw_pointer_cast(tmp.data()),
            thrust::raw_pointer_cast(lse.data()), num_qo_heads, num_kv_heads,
            qo_len, kv_len, head_dim, kv_layout, pos_encoding_mode, causal,
            use_fp16_qk_reduction, sm_scale, window_left, rope_scale,
            rope_theta, stream);

        if (status != hipSuccess) {
            std::cerr << "Error during warmup: " << hipGetErrorString(status)
                      << std::endl;
            return 1;
        }
    }

    // Timing runs
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, stream);

    for (int i = 0; i < iterations; ++i) {
        hipError_t status = dispatch_single_prefill(
            thrust::raw_pointer_cast(q.data()),
            thrust::raw_pointer_cast(k.data()),
            thrust::raw_pointer_cast(v.data()),
            thrust::raw_pointer_cast(o.data()),
            thrust::raw_pointer_cast(tmp.data()),
            thrust::raw_pointer_cast(lse.data()), num_qo_heads, num_kv_heads,
            qo_len, kv_len, head_dim, kv_layout, pos_encoding_mode, causal,
            use_fp16_qk_reduction, sm_scale, window_left, rope_scale,
            rope_theta, stream);

        if (status != hipSuccess) {
            std::cerr << "Error during benchmark: "
                      << hipGetErrorString(status) << std::endl;
            return 1;
        }
    }

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);

    float elapsed_ms;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    float avg_ms = elapsed_ms / iterations;

    // Calculate FLOPS
    double flops =
        calculate_flops(qo_len, kv_len, num_qo_heads, head_dim, causal);
    double tflops = flops / (avg_ms * 1e-3) / 1e12;

    // Report results
    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Performance Results:" << std::endl;
    std::cout << "  Average time: " << avg_ms << " ms" << std::endl;
    std::cout << "  Performance: " << tflops << " TFLOPS" << std::endl;

    // Run validation if requested
    if (validate) {
        std::cout << "\nRunning validation..." << std::endl;

        // Copy output from GPU to host for validation
        thrust::host_vector<half> h_output = o;

        // Create input data on host for CPU reference
        std::vector<half> h_q(q.begin(), q.end());
        std::vector<half> h_k(k.begin(), k.end());
        std::vector<half> h_v(v.begin(), v.end());

        // Compute reference output on CPU
        std::vector<half> ref_output = reference::single_mha(
            h_q, h_k, h_v, qo_len, kv_len, num_qo_heads, num_kv_heads, head_dim,
            causal, kv_layout, pos_encoding_mode, rope_scale, rope_theta);

        // Validate results
        bool validation_passed = validate_results(h_output, ref_output);

        // Report validation status
        std::cout << "Validation " << (validation_passed ? "PASSED" : "FAILED")
                  << std::endl;
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);

    return 0;
}
